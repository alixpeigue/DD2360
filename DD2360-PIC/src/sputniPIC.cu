/** A mixed-precision implicit Particle-in-Cell simulator for heterogeneous systems **/

#include <hip/hip_runtime.h>

// Allocator for 2D, 3D and 4D array: chain of pointers
#include "Alloc.h"

// Precision: fix precision for different quantities
#include "PrecisionTypes.h"
// Simulation Parameter - structure
#include "Parameters.h"
// Grid structure
#include "Grid.h"
// Interpolated Quantities Structures
#include "InterpDensSpecies.h"
#include "InterpDensNet.h"

// Field structure
#include "EMfield.h" // Just E and Bn
#include "EMfield_aux.h" // Bc, Phi, Eth, D

// Particles structure
#include "Particles.h"
#include "Particles_aux.h" // Needed only if dointerpolation on GPU - avoid reduction on GPU

// Initial Condition
#include "IC.h"
// Boundary Conditions
#include "BC.h"
// timing
#include "Timing.h"
// Read and output operations
#include "RW_IO.h"


int main(int argc, char **argv){
    
    // Read the inputfile and fill the param structure
    parameters param;
    // Read the input file name from command line
    readInputFile(&param,argc,argv);
    printParameters(&param);
    saveParameters(&param);
    
    // Timing variables
    double iStart = cpuSecond();
    double iMover, iInterp, eMover = 0.0, eInterp= 0.0;
    
    // Set-up the grid information
    grid grd;
    setGrid(&param, &grd);
    
    // Allocate Fields
    EMfield field;
    field_allocate(&grd,&field);
    EMfield_aux field_aux;
    field_aux_allocate(&grd,&field_aux);
    
    
    // Allocate Interpolated Quantities
    // per species
    interpDensSpecies *ids = new interpDensSpecies[param.ns];
    for (int is=0; is < param.ns; is++)
        interp_dens_species_allocate(&grd,&ids[is],is);
    // Net densities
    interpDensNet idn;
    interp_dens_net_allocate(&grd,&idn);
    
    // Allocate Particles
    particles *part = new particles[param.ns];
    // allocation
    for (int is=0; is < param.ns; is++){
        particle_allocate(&param,&part[is],is);
    }
    
    // Initialization
    initGEM(&param,&grd,&field,&field_aux,part,ids);
    
    
    // **********************************************************//
    // **** Start the Simulation!  Cycle index start from 1  *** //
    // **********************************************************//
    for (int cycle = param.first_cycle_n; cycle < (param.first_cycle_n + param.ncycles); cycle++) {
        
        std::cout << std::endl;
        std::cout << "***********************" << std::endl;
        std::cout << "   cycle = " << cycle << std::endl;
        std::cout << "***********************" << std::endl;
    
        // set to zero the densities - needed for interpolation
        setZeroDensities(&idn,ids,&grd,param.ns);
        
        
        
        // implicit mover
        // iMover = cpuSecond(); // start timer for mover
        // for (int is=0; is < param.ns; is++)
        //     mover_PC(&part[is],&field,&grd,&param);
        // eMover += (cpuSecond() - iMover); // stop timer for mover
        
        iMover = cpuSecond(); // start timer for mover
        FPfield *grd_XN_flat, *grd_YN_flat, *grd_ZN_flat, *field_Ex_flat, *field_Ey_flat, *field_Ez_flat, *field_Bxn_flat, *field_Byn_flat, *field_Bzn_flat;

        hipMalloc(&grd_XN_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn);
        hipMalloc(&grd_YN_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn);
        hipMalloc(&grd_ZN_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn);
        hipMalloc(&field_Ex_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn);
        hipMalloc(&field_Ey_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn);
        hipMalloc(&field_Ez_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn);
        hipMalloc(&field_Bxn_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn);
        hipMalloc(&field_Byn_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn);
        hipMalloc(&field_Bzn_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn);

        hipMemcpy(grd_XN_flat, grd.XN_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn, hipMemcpyHostToDevice);
        hipMemcpy(grd_YN_flat, grd.YN_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn, hipMemcpyHostToDevice);
        hipMemcpy(grd_ZN_flat, grd.ZN_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn, hipMemcpyHostToDevice);
        hipMemcpy(field_Ex_flat, field.Ex_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn, hipMemcpyHostToDevice);
        hipMemcpy(field_Ey_flat, field.Ey_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn, hipMemcpyHostToDevice);
        hipMemcpy(field_Ez_flat, field.Ez_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn, hipMemcpyHostToDevice);
        hipMemcpy(field_Bxn_flat, field.Bxn_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn, hipMemcpyHostToDevice);
        hipMemcpy(field_Byn_flat, field.Byn_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn, hipMemcpyHostToDevice);
        hipMemcpy(field_Bzn_flat, field.Bzn_flat, sizeof(FPfield) * grd.nxn * grd.nyn * grd.nzn, hipMemcpyHostToDevice);

        
        for (int is=0; is < param.ns; is++) {
            std::cout << "***  MOVER with SUBCYCLYING "<< param.n_sub_cycles << " - species " << part[is].species_ID << " ***" << std::endl;
            FPpart *part_x, *part_y, *part_z, *part_u, *part_v, *part_w;

            hipMalloc(&part_x, sizeof(FPpart) * part[is].npmax);
            hipMalloc(&part_y, sizeof(FPpart) * part[is].npmax);
            hipMalloc(&part_z, sizeof(FPpart) * part[is].npmax);
            hipMalloc(&part_u, sizeof(FPpart) * part[is].npmax);
            hipMalloc(&part_v, sizeof(FPpart) * part[is].npmax);
            hipMalloc(&part_w, sizeof(FPpart) * part[is].npmax);

            hipMemcpy(part_x, part[is].x, sizeof(FPpart) * part[is].npmax, hipMemcpyHostToDevice);
            hipMemcpy(part_y, part[is].y, sizeof(FPpart) * part[is].npmax, hipMemcpyHostToDevice);
            hipMemcpy(part_z, part[is].z, sizeof(FPpart) * part[is].npmax, hipMemcpyHostToDevice);
            hipMemcpy(part_u, part[is].u, sizeof(FPpart) * part[is].npmax, hipMemcpyHostToDevice);
            hipMemcpy(part_v, part[is].v, sizeof(FPpart) * part[is].npmax, hipMemcpyHostToDevice);
            hipMemcpy(part_w, part[is].w, sizeof(FPpart) * part[is].npmax, hipMemcpyHostToDevice);

            int threadsPerBlock = 256;
            int blocksPerGrid = (part[is].nop + threadsPerBlock - 1) / threadsPerBlock;

            mover_PC_gpu<<<blocksPerGrid, threadsPerBlock>>>(part[is],field,grd,param, part_x, part_y, part_z, part_u, part_v, part_w, grd_XN_flat, grd_YN_flat,
                grd_ZN_flat, field_Ex_flat, field_Ey_flat, field_Ez_flat, field_Bxn_flat, field_Byn_flat, field_Bzn_flat);

            hipMemcpy(part[is].x, part_x, sizeof(FPpart) * part[is].npmax, hipMemcpyDeviceToHost);
            hipMemcpy(part[is].y, part_y, sizeof(FPpart) * part[is].npmax, hipMemcpyDeviceToHost);
            hipMemcpy(part[is].z, part_z, sizeof(FPpart) * part[is].npmax, hipMemcpyDeviceToHost);
            hipMemcpy(part[is].u, part_u, sizeof(FPpart) * part[is].npmax, hipMemcpyDeviceToHost);
            hipMemcpy(part[is].v, part_v, sizeof(FPpart) * part[is].npmax, hipMemcpyDeviceToHost);
            hipMemcpy(part[is].w, part_w, sizeof(FPpart) * part[is].npmax, hipMemcpyDeviceToHost);

            hipFree(part_x);
            hipFree(part_y);
            hipFree(part_z);
            hipFree(part_u);
            hipFree(part_v);
            hipFree(part_w);
        }


        hipFree(grd_XN_flat);
        hipFree(grd_YN_flat);
        hipFree(grd_ZN_flat);
        hipFree(field_Ex_flat);
        hipFree(field_Ey_flat);
        hipFree(field_Ez_flat);
        hipFree(field_Bxn_flat);
        hipFree(field_Byn_flat);
        hipFree(field_Bzn_flat);

        
        eMover += (cpuSecond() - iMover); // stop timer for mover
        
        // interpolation particle to grid
        iInterp = cpuSecond(); // start timer for the interpolation step
        // interpolate species
        for (int is=0; is < param.ns; is++)
            interpP2G(&part[is],&ids[is],&grd);
        // apply BC to interpolated densities
        for (int is=0; is < param.ns; is++)
            applyBCids(&ids[is],&grd,&param);
        // sum over species
        sumOverSpecies(&idn,ids,&grd,param.ns);
        // interpolate charge density from center to node
        applyBCscalarDensN(idn.rhon,&grd,&param);
        
        
        
        // write E, B, rho to disk
        if (cycle%param.FieldOutputCycle==0){
            VTK_Write_Vectors(cycle, &grd,&field);
            VTK_Write_Scalars(cycle, &grd,ids,&idn);
        }
        
        eInterp += (cpuSecond() - iInterp); // stop timer for interpolation
        
        
    
    }  // end of one PIC cycle
    
    /// Release the resources
    // deallocate field
    grid_deallocate(&grd);
    field_deallocate(&grd,&field);
    // interp
    interp_dens_net_deallocate(&grd,&idn);
    
    // Deallocate interpolated densities and particles
    for (int is=0; is < param.ns; is++){
        interp_dens_species_deallocate(&grd,&ids[is]);
        particle_deallocate(&part[is]);
    }
    
    
    // stop timer
    double iElaps = cpuSecond() - iStart;
    
    // Print timing of simulation
    std::cout << std::endl;
    std::cout << "**************************************" << std::endl;
    std::cout << "   Tot. Simulation Time (s) = " << iElaps << std::endl;
    std::cout << "   Mover Time / Cycle   (s) = " << eMover/param.ncycles << std::endl;
    std::cout << "   Interp. Time / Cycle (s) = " << eInterp/param.ncycles  << std::endl;
    std::cout << "**************************************" << std::endl;
    
    // exit
    return 0;
}


